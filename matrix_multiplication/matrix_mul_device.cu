
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

__global__
void MatrixMultiplication(unsigned long* Md, unsigned long* Nd, unsigned long* Pd, int width) {
    int tx = threadIdx.x; // x-index of threads
    int ty = threadIdx.y; // y-index of threads

    // Pvalue stores the Pd element that is computed by thread
    unsigned long Pvalue = 0;

    for(int k = 0; k < width; k++) {
        unsigned long Mdelement = Md[ty * width + k];
        unsigned long Ndelement = Nd[k * width + tx];
        Pvalue += Mdelement * Ndelement;
    }

    // Write the matrix to device memory
    // Each thread write one element
    Pd[ty * width + tx] = Pvalue;
}

int main(int argc, char **argv) {


    int width = 1500;
    
    cout << width << " x " << width << endl;
    
    unsigned long size = width * width * sizeof(unsigned long);
    unsigned long* A;
    unsigned long* B;
    unsigned long* C;

    A = (unsigned long*) malloc(size);
    B = (unsigned long*) malloc(size);
    C = (unsigned long*) malloc(size);
    srand (time(NULL));
    
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            A[i * width + j] = (rand() % 100) + 1;
            B[i * width + j] = (rand() % 200) + 1;
        }
    }

    unsigned long* Ad;
    unsigned long* Bd;
    unsigned long* Cd;

    // transfer A and B to device memory
    hipMalloc((void**) &Ad, size);
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &Bd, size);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

    // allocate c on the device
    // C is a result of matrix multiplication
    hipMalloc((void**) &Cd, size);

    dim3 dimBlock(width, width);
    dim3 dimGrid(1,1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    MatrixMultiplication<<<dimGrid,dimBlock>>>(Ad, Bd, Cd, width);
    hipEventRecord(stop);

    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << endl;
    cout << milliseconds << endl;

    // for (int i = 0; i < width; i++) {
    //     for (int j = 0; j < width; j++) {
    //         cout << C[i * width + j] << "\t";
    //     }
    //     cout << endl;
    // }

    return 0;
}