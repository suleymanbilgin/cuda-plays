
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <ctime> 

using namespace std;

void matrix_multiplication(float* M, float* N, float* P, int witdh) {
    for (int i = 0; i < witdh; i++) {
        for (int j = 0; j < witdh; j++) {
            int sum = 0;
            for (int k = 0; k < witdh; k++) {
                float a = M[i * witdh + k];
                float b = N[k * witdh + j];
                
                sum += a * b;
            }
            P[i * witdh + j] = sum;
        }
    }
}

int main(int argc, char **argv) {
    int witdh = 200;

    float* A;
    float* B;
    float* C;

    A = (float*) malloc(witdh * witdh * sizeof(float));
    B = (float*) malloc(witdh * witdh * sizeof(float));
    C = (float*) malloc(witdh * witdh * sizeof(float));
    srand (time(NULL));

    // for (int i = 0; i < witdh; i++) {
    //     for (int j = 0; j < witdh; j++) {
    //         A[i * witdh + j] = (rand() % 100) + 1;
    //         B[i * witdh + j] = (rand() % 200) + 1;
    //     }
    // }
    // cout << endl;
    // for (int i = 0; i < witdh; i++) {
    //     for (int j = 0; j < witdh; j++) {
    //         cout << A[i * witdh + j] << "\t";
    //     }
    //     cout << endl;
    // }

    // cout << endl;

    // for (int i = 0; i < witdh; i++) {
    //     for (int j = 0; j < witdh; j++) {
    //         cout << B[i * witdh + j] << "\t";
    //     }
    //     cout << endl;
    // }

    matrix_multiplication(A, B, C, witdh);

    /*for (int i = 0; i < witdh; i++) {
        for (int j = 0; j < witdh; j++) {
            cout << C[i * witdh + j] << "\t";
        }
        cout << endl;
    }*/

    return 0;
}
